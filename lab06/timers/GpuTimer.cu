//
// Created by xgg on 2 Jun 2022.
//

#include "GpuTimer.cuh"

struct PrivateTimingGPU {
    hipEvent_t     _start;
    hipEvent_t     _stop;
};

GpuTimer::GpuTimer() {
    privateTimingGPU = new PrivateTimingGPU;

    hipEventCreate(&(*privateTimingGPU)._start);
    hipEventCreate(&(*privateTimingGPU)._stop);
}

GpuTimer::~GpuTimer() {
    hipEventDestroy((*privateTimingGPU)._start);
    hipEventDestroy((*privateTimingGPU)._stop);
}

void GpuTimer::start() {
    hipEventRecord((*privateTimingGPU)._start, nullptr);
}

void GpuTimer::stop() {
    hipEventRecord((*privateTimingGPU)._stop, nullptr);
}

float GpuTimer::elapsed() {
    float elapsed;
    hipEventSynchronize((*privateTimingGPU)._stop);
    hipEventElapsedTime(&elapsed, (*privateTimingGPU)._start, (*privateTimingGPU)._stop);
    return elapsed;
}
