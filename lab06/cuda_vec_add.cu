#include "hip/hip_runtime.h"
//
// Created by xgg on 2 Jun 2022.
//
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <utility>
#include "Utility.h"
#include "GpuTimer.h"


// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(const double *a, const double *b, double *c, const int n) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

namespace gpu {
    std::pair<float, int> vec_add(const int n, const int block_size) {
        GpuTimer gpuTimer;

        // Host input vectors
        double *h_a;
        double *h_b;
        // Host output vector
        double *h_c;

        // Device input vectors
        double *d_a;
        double *d_b;
        //Device output vector
        double *d_c;

        // Size, in bytes, of each vector
        size_t bytes = n * sizeof(double);

        // Allocate memory for each vector on host
        h_a = (double *) malloc(bytes);
        h_b = (double *) malloc(bytes);
        h_c = (double *) malloc(bytes);

        // Allocate memory for each vector on GPU
        hipMalloc(&d_a, bytes);
        hipMalloc(&d_b, bytes);
        hipMalloc(&d_c, bytes);

        int i;
        // Initialize vectors on host
        for (i = 0; i < n; i++) {
            h_a[i] = sin(i) * sin(i);
            h_b[i] = cos(i) * cos(i);
        }

        // Start time measurement
        gpuTimer.start();

        // Copy host vectors to device
        hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

        int gridSize;

        // Number of thread blocks in grid
        gridSize = (int) ceil((float) n / block_size);

        // Execute the kernel
        vecAdd<<<gridSize, block_size>>>(d_a, d_b, d_c, n);

        // Copy array back to host
        hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

        gpuTimer.stop();

        int result = Utility::check_result(h_c, n);

        // Release device memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        // Release host memory
        free(h_a);
        free(h_b);
        free(h_c);

        return std::pair<float, int>{gpuTimer.elapsed(), result};
    }
}