//
// Created by xgg on 2 Jun 2022.
//

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>


// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(const double *a, const double *b, double *c, const int n) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

namespace gpu {
    int vec_add(const double *h_a, const double *h_b, double *h_c, const int n, const int threads) {

        // Size, in bytes, of each vector
        size_t bytes = n * sizeof(double);

        // Device input vectors
        double *d_a;
        double *d_b;
        //Device output vector
        double *d_c;

        // Allocate memory for each vector on GPU
        hipMalloc(&d_a, bytes);
        hipMalloc(&d_b, bytes);
        hipMalloc(&d_c, bytes);

        // Copy host vectors to device
        hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

        int gridSize;

        // Number of thread blocks in grid
        gridSize = (int) ceil((float) n / threads);

        // Execute the kernel
        vecAdd<<<gridSize, threads>>>(d_a, d_b, d_c, n);

        // Copy array back to host
        hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

        // Release device memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        return 0;
    }
}