//
// Created by xgg on 2 Jun 2022.
//

#include "GpuTimer.h"

GpuTimer::GpuTimer() {
    hipEventCreate(&_start);
    hipEventCreate(&_stop);
}

GpuTimer::~GpuTimer() {
    hipEventDestroy(_start);
    hipEventDestroy(_stop);
}

void GpuTimer::start() {
    hipEventRecord(_start, 0);
}

void GpuTimer::stop() {
    hipEventRecord(_stop, 0);
}

float GpuTimer::elapsed() {
    float elapsed;
    hipEventSynchronize(_stop);
    hipEventElapsedTime(&elapsed, _start, _stop);
    return elapsed;
}
